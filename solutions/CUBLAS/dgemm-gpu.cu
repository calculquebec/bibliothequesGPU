// This example computes real matrix C=alpha*A*B+beta*C using Intel® MKL function dgemm, 
// where A, B, and  C are matrices and alpha and beta are double precision scalars

#define min(x,y) (((x) < (y)) ? (x) : (y))


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

int main()
{
    // Define variables
    double *A, *B, *C;
    int m, n, k, i, j;
    double alpha, beta;

    double *d_A, *d_B, *d_C;

    // Set up the matrices
    m = 2000, k = 2000, n = 2000;
    printf (" Initializing data for matrix multiplication C=A*B for matrix \n"
            " A(%ix%i) and matrix B(%ix%i)\n\n", m, k, k, n);
    alpha = 1.0; beta = 0.0;

    // Allocate CPU memory
    printf (" Allocating memory for matrices aligned on 64-byte boundary for better \n"
            " performance \n\n");
    A = (double *)malloc( m*k*sizeof( double ) );
    B = (double *)malloc( k*n*sizeof( double ) );
    C = (double *)malloc( m*n*sizeof( double ) );

    // Set GPU device
    hipSetDevice(0);

    // Allocate GPU memory
    hipMalloc((void**)&d_A, m*k*sizeof(double));
    hipMalloc((void**)&d_B, k*n*sizeof(double));
    hipMalloc((void**)&d_C, m*n*sizeof(double));

    // Initialize matrices
    printf (" Intializing matrix data \n\n");
    for (i = 0; i < (m*k); i++) {
        A[i] = (double)(i+1);
    }

    for (i = 0; i < (k*n); i++) {
        B[i] = (double)(-i-1);
    }

    for (i = 0; i < (m*n); i++) {
        C[i] = 0.0;
    }

    /* Copy data to GPU  */
    hipblasSetVector(m*k, sizeof(double), A, 1, d_A, 1);
    hipblasSetVector(k*n, sizeof(double), B, 1, d_B, 1);


    /* Initialize cuBLAS */
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    /* Perform multiplication on GPU  */
    for(int i=0;i<20;i++) hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,m,n,k,&alpha, d_A, m, d_B, k, &beta, d_C, m); 


    /* Finalize cuBLAS  */
    hipblasDestroy(handle);


    /* Copy results back to CPU */
    hipblasGetVector(m*n, sizeof(double), d_C, 1, C, 1);

    /* Print the results */
    printf ("\n Computations completed.\n\n");
    printf (" Top left corner of matrix A: \n");
    for (i=0; i<min(m,6); i++) {
      for (j=0; j<min(k,6); j++) {
        printf ("%12.0f", A[j+i*k]);
      }
      printf ("\n");
    }

    printf ("\n Top left corner of matrix B: \n");
    for (i=0; i<min(k,6); i++) {
      for (j=0; j<min(n,6); j++) {
        printf ("%12.0f", B[j+i*n]);
      }
      printf ("\n");
    }
    
    printf ("\n Top left corner of matrix C: \n");
    for (i=0; i<min(m,6); i++) {
      for (j=0; j<min(n,6); j++) {
        printf ("%12.5G", C[j+i*n]);
      }
      printf ("\n");
    }

    // Free CPU memory
    printf ("\n Deallocating memory \n\n");
    free(A);
    free(B);
    free(C);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf (" Example completed. \n\n");
    return 0;
}
