#define NX 256


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>



int main(){

   /* Define FFT variables */
   hipfftComplex *in, *out;
   hipfftHandle plan;

   /* Set the GPU device */
   hipSetDevice(0);

   /* Allocate memory on GPU for FFT data */
   hipMalloc((void**)&in, NX*sizeof(hipfftComplex));
   hipMalloc((void**)&out, NX*sizeof(hipfftComplex));


   /* Create FFT plan */
   hipfftPlan1d(&plan, NX, HIPFFT_C2C, 1);

   /* Perform complex-to-complex FFT transformation */
   hipfftExecC2C(plan, in, out, HIPFFT_FORWARD);
   printf("Checking error: %s\n",hipGetErrorString(hipGetLastError()));

   /* Destroy FFT plan */
   hipfftDestroy(plan);

   /* Free the memory */
   hipFree(in);
   hipFree(out);

}
